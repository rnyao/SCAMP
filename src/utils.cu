#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include "utils.h"
namespace SCAMP {

// This kernel computes a sliding mean with specified window size and a
// corresponding prefix sum array (A)
__global__ void sliding_mean(double *pref_sum, size_t window, size_t size,
                             double *means) {
  const double coeff = 1.0 / (double)window;
  size_t a = blockIdx.x * blockDim.x + threadIdx.x;
  size_t b = blockIdx.x * blockDim.x + threadIdx.x + window;

  if (a == 0) {
    means[a] = pref_sum[window - 1] * coeff;
  }
  if (a < size - 1) {
    means[a + 1] = (pref_sum[b] - pref_sum[a]) * coeff;
  }
}

__global__ void sliding_norm(double *cumsumsqr, unsigned int window,
                             unsigned int size, double *norms) {
  int a = blockIdx.x * blockDim.x + threadIdx.x;
  int b = blockIdx.x * blockDim.x + threadIdx.x + window;
  if (a == 0) {
    norms[a] = 1 / sqrt(cumsumsqr[window - 1]);
  } else if (b < size + window) {
    norms[a] = 1 / sqrt(cumsumsqr[b - 1] - cumsumsqr[a - 1]);
  }
}

__global__ void sliding_dfdg(const double *T, const double *means, double *df,
                             double *dg, const int m, const int n) {
  const double half = 1.0 / (double)2.0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n - 1) {
    df[tid] = (T[tid + m] - T[tid]) * half;
    dg[tid] = (T[tid + m] - means[tid + 1]) + (T[tid] - means[tid]);
  }
}

__global__ void __launch_bounds__(512, 4)
    fastinvnorm(double *norm, const double *mean, const double *T, int m,
                int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int jump = ceil(n / (double)(blockDim.x * gridDim.x));
  int start = jump * tid;
  int end = jump * (tid + 1);
  end = min(end, n);
  if (start >= n) {
    return;
  }
  double sum = 0;
  for (int i = 0; i < m; ++i) {
    double val = T[i + start] - mean[start];
    sum += val * val;
  }
  norm[start] = sum;

  for (int i = start + 1; i < end; ++i) {
    norm[i] =
        norm[i - 1] + ((T[i - 1] - mean[i - 1]) + (T[i + m - 1] - mean[i])) *
                          (T[i + m - 1] - T[i - 1]);
  }
  for (int i = start; i < end; ++i) {
    norm[i] = 1.0 / sqrt(norm[i]);
  }
}

__global__ void cross_correlation_to_ed(float *profile, unsigned int n,
                                        unsigned int m) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    profile[tid] = sqrt(max(2 * (1 - profile[tid]), 0.0)) * sqrt((double)m);
  }
}

__global__ void merge_mp_idx(float *mp, uint32_t *mpi, uint32_t n,
                             uint64_t *merged) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    mp_entry item;
    item.floats[0] = (float)mp[tid];
    item.ints[1] = mpi[tid];
    merged[tid] = item.ulong;
  }
}

void elementwise_max_with_index(std::vector<float> &mp_full,
                                std::vector<uint32_t> &mpi_full,
                                int64_t merge_start, int64_t tile_sz,
                                std::vector<uint64_t> *to_merge) {
  for (int i = 0; i < tile_sz; ++i) {
    mp_entry curr;
    curr.ulong = to_merge->at(i);
    if (mp_full[i + merge_start] < curr.floats[0]) {
      mp_full[i + merge_start] = curr.floats[0];
      mpi_full[i + merge_start] = curr.ints[1];
    }
  }
}

void compute_statistics(const double *T, double *norms, double *df, double *dg,
                        double *means, size_t n, size_t m, hipStream_t s,
                        double *scratch) {
  dim3 grid(ceil(n / (double)512), 1, 1);
  dim3 block(512, 1, 1);
  thrust::device_ptr<const double> dev_ptr_T =
      thrust::device_pointer_cast<const double>(T);
  thrust::device_ptr<double> dev_ptr_scratch =
      thrust::device_pointer_cast<double>(scratch);
  thrust::inclusive_scan(thrust::cuda::par.on(s), dev_ptr_T,
                         dev_ptr_T + n + m - 1, dev_ptr_scratch,
                         thrust::plus<double>());
  // hipcub::DeviceScan::InclusiveSum(temp, bytes, T, scratch, n + m - 1, s);
  // Allocate temporary storage
  // hipMalloc(&temp, bytes);
  // hipcub::DeviceScan::InclusiveSum(temp, bytes, T, scratch, n + m - 1, s);
  // hipFree(temp);
  // prefix_sum(T, n+m-1, scratch, s);
  gpuErrchk(hipPeekAtLastError());
  // Use prefix sum to compute sliding mean
  sliding_mean<<<grid, block, 0, s>>>(scratch, m, n, means);
  gpuErrchk(hipPeekAtLastError());

  // Compute differential values
  sliding_dfdg<<<grid, block, 0, s>>>(T, means, df, dg, m, n);
  gpuErrchk(hipPeekAtLastError());

  // This will be kind of slow on the GPU, may cause latency between tiles
  int workers = n / m + 1;
  fastinvnorm<<<dim3(ceil(workers / (double)512), 1, 1), dim3(512, 1, 1), 0,
                s>>>(norms, means, T, m, n);
  gpuErrchk(hipPeekAtLastError());
}

void launch_merge_mp_idx(float *mp, uint32_t *mpi, uint32_t n, uint64_t *merged,
                         hipStream_t s) {
  merge_mp_idx<<<dim3(std::ceil(n / 1024.0), 1, 1), dim3(1024, 1, 1), 0, s>>>(
      mp, mpi, n, merged);
}

}  // namespace SCAMP
